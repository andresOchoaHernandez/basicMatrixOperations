
#include <hip/hip_runtime.h>
int main(void)
{
    int deviceCount;
    hipError_t e = hipGetDeviceCount(&deviceCount);
    return e == hipSuccess ? deviceCount : -1;
}