#include "hip/hip_runtime.h"
#include <iostream>

#include "BasicMatrixOperations.cuh"

__global__
void matrix_multiplication_kernel(const double *A,const int A_columns_B_rows,const double *B,double *C,const int C_rows,const int C_columns)
{
    int row    = blockIdx.y * blockDim.y + threadIdx.y;
    int column = blockIdx.x * blockDim.x + threadIdx.x;

    if(row >= C_rows || column >= C_columns) return;

    double sum = 0;
    for(int k = 0; k < A_columns_B_rows;k++)
    {
        sum += A[row * A_columns_B_rows + k] * B[k * C_columns + column]; 
    }
 
    C[row * C_columns + column] = sum;
}
__host__
int gpu_matrix_multiplication(const Matrix2d& A, const Matrix2d& B,Matrix2d& C)
{
    if( (A.columns != B.rows) || (A.rows != C.rows) || ( B.columns != C.columns) )
    {
        std::cerr << "Given matrices' dimensions don't match:"           << std::endl
                  << "param : A (" << A.rows <<" x " << A.columns << ")" << std::endl
                  << "param : B (" << B.rows <<" x " << B.columns << ")" << std::endl
                  << "param : C (" << C.rows <<" x " << C.columns << ")" << std::endl;
        return -1;
    }

    double *d_A,*d_B,*d_C;

    hipMalloc(&d_A,static_cast<size_t>(A.rows * A.columns * sizeof(double)));
    hipMalloc(&d_B,static_cast<size_t>(B.rows * B.columns * sizeof(double)));
    hipMalloc(&d_C,static_cast<size_t>(C.rows * C.columns * sizeof(double)));

    hipMemcpy(d_A,A.data,static_cast<size_t>(A.rows * A.columns * sizeof(double)),hipMemcpyHostToDevice);
    hipMemcpy(d_B,B.data,static_cast<size_t>(B.rows * B.columns * sizeof(double)),hipMemcpyHostToDevice);

    dim3 blockDim(32,32,1);
    dim3 gridDim((C.columns + 32 -1)/32 + 1,(C.rows + 32 -1)/32 + 1,1);

    matrix_multiplication_kernel<<<gridDim,blockDim>>>(d_A,A.columns,d_B,d_C,C.rows,C.columns);

    hipDeviceSynchronize();

    hipMemcpy(C.data,d_C,static_cast<size_t>(C.rows * C.columns * sizeof(double)),hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipDeviceReset();

    return 1;
}

__global__
void matrix_transpose_kernel(const double *A,double *A_t,const int A_rows,const int A_columns)
{
    int row    = blockIdx.y * blockDim.y + threadIdx.y; 
    int column = blockIdx.x * blockDim.x + threadIdx.x;

    if(row >= A_rows || column >= A_columns) return;

    A_t[column*A_rows + row] = A[row*A_columns + column];
}

__host__
int gpu_matrix_transpose(const Matrix2d& A,Matrix2d& A_t)
{
    if( (A.rows != A_t.columns) || (A.columns != A_t.rows) )
    {
        std::cerr << "Given matrices' dimensions don't match:"                 << std::endl
                  << "param A  : (" << A.rows << " x " << A.columns << ")"     << std::endl
                  << "param A_t: (" << A_t.rows << " x " << A_t.columns << ")" << std::endl;
        return -1;
    }

    double *d_A,*d_A_t;

    hipMalloc(&d_A,static_cast<size_t>(A.rows * A.columns * sizeof(double)));
    hipMalloc(&d_A_t,static_cast<size_t>(A_t.rows * A_t.columns * sizeof(double)));

    hipMemcpy(d_A,A.data,static_cast<size_t>(A.rows * A.columns * sizeof(double)),hipMemcpyHostToDevice);

    dim3 blockDim(32,32,1);
    dim3 gridDim((A.columns + 32 -1)/32 + 1,(A.rows + 32 -1)/32 + 1,1);

    matrix_transpose_kernel<<<gridDim,blockDim>>>(d_A,d_A_t,A.rows,A.columns);

    hipDeviceSynchronize();

    hipMemcpy(A_t.data,d_A_t,static_cast<size_t>(A_t.rows * A_t.columns * sizeof(double)),hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_A_t);

    hipDeviceReset();

    return 1;
}


__global__
void matrix_dot_product_kernel(const double *A, const double *B,double *C, const int C_rows,const int C_columns)
{
    int row    = blockIdx.y * blockDim.y + threadIdx.y; 
    int column = blockIdx.x * blockDim.x + threadIdx.x;

    if(row >= C_rows || column >= C_columns) return;

    C[row * C_columns + column] = A[row*C_columns+column] * B[row*C_columns+column];
}

__host__
int gpu_matrix_dot_product(const Matrix2d& A, const Matrix2d& B,Matrix2d& C)
{
    if(
        !(            
            A.rows    == B.rows     && 
            A.columns == B.columns  &&
            C.rows    == A.rows     &&
            C.columns == A.columns
         )
      )
    {

        std::cerr << "Given matrices' dimensions don't match:"           << std::endl
                  << "param : A (" << A.rows <<" x " << A.columns << ")" << std::endl
                  << "param : B (" << B.rows <<" x " << B.columns << ")" << std::endl
                  << "param : C (" << C.rows <<" x " << C.columns << ")" << std::endl;
        return -1;
    }

    double *d_A,*d_B,*d_C;

    hipMalloc(&d_A,static_cast<size_t>(A.rows * A.columns * sizeof(double)));
    hipMalloc(&d_B,static_cast<size_t>(B.rows * B.columns * sizeof(double)));
    hipMalloc(&d_C,static_cast<size_t>(C.rows * C.columns * sizeof(double)));

    hipMemcpy(d_A,A.data,static_cast<size_t>(A.rows * A.columns * sizeof(double)),hipMemcpyHostToDevice);
    hipMemcpy(d_B,B.data,static_cast<size_t>(B.rows * B.columns * sizeof(double)),hipMemcpyHostToDevice);

    dim3 blockDim(32,32,1);
    dim3 gridDim((C.columns + 32 -1)/32 + 1,(C.rows + 32 -1)/32 + 1,1);

    matrix_dot_product_kernel<<<gridDim,blockDim>>>(d_A,d_B,d_C,C.rows,C.columns);

    hipDeviceSynchronize();

    hipMemcpy(C.data,d_C,static_cast<size_t>(C.rows * C.columns * sizeof(double)),hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipDeviceReset();

    return 1;
}


__global__
void scalar_matrix_dot_product_kernel(const double scalar, const double *A,double *C, const int C_rows,const int C_columns)
{
    int row    = blockIdx.y * blockDim.y + threadIdx.y; 
    int column = blockIdx.x * blockDim.x + threadIdx.x;

    if(row >= C_rows || column >= C_columns) return;

    C[row * C_columns + column] = scalar * A[row*C_columns+column];
}

__host__
int gpu_scalar_matrix_dot_product(const double scalar, const Matrix2d& A,Matrix2d& C)
{
    if(!(C.rows    == A.rows && C.columns == A.columns))
    {
        std::cerr << "Given matrices' dimensions don't match:"           << std::endl
                  << "param : A (" << A.rows <<" x " << A.columns << ")" << std::endl
                  << "param : C (" << C.rows <<" x " << C.columns << ")" << std::endl;
        return -1;
    }

    double *d_A,*d_C;

    hipMalloc(&d_A,static_cast<size_t>(A.rows * A.columns * sizeof(double)));
    hipMalloc(&d_C,static_cast<size_t>(C.rows * C.columns * sizeof(double)));

    hipMemcpy(d_A,A.data,static_cast<size_t>(A.rows * A.columns * sizeof(double)),hipMemcpyHostToDevice);

    dim3 blockDim(32,32,1);
    dim3 gridDim((C.columns + 32 -1)/32 + 1,(C.rows + 32 -1)/32 + 1,1);

    scalar_matrix_dot_product_kernel<<<gridDim,blockDim>>>(scalar,d_A,d_C,C.rows,C.columns);

    hipDeviceSynchronize();

    hipMemcpy(C.data,d_C,static_cast<size_t>(C.rows * C.columns * sizeof(double)),hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_C);

    hipDeviceReset();

    return 1;
}
__global__
void matrix_sum_kernel(const double *A,const double *B, double *C,const int C_rows,const int C_columns)
{
    int row    = blockIdx.y * blockDim.y + threadIdx.y; 
    int column = blockIdx.x * blockDim.x + threadIdx.x;

    if(row >= C_rows || column >= C_columns) return;

    C[row * C_columns + column] = A[row*C_columns+column] + B[row*C_columns+column];
}

__host__
int gpu_matrix_sum(const Matrix2d& A,const Matrix2d& B, Matrix2d& C)
{
    if(
        !(            
            A.rows    == B.rows     && 
            A.columns == B.columns  &&
            C.rows    == A.rows     &&
            C.columns == A.columns
         )
      )
    {

        std::cerr << "Given matrices' dimensions don't match:"           << std::endl
                  << "param : A (" << A.rows <<" x " << A.columns << ")" << std::endl
                  << "param : B (" << B.rows <<" x " << B.columns << ")" << std::endl
                  << "param : C (" << C.rows <<" x " << C.columns << ")" << std::endl;
        return -1;
    }

    double *d_A,*d_B,*d_C;

    hipMalloc(&d_A,static_cast<size_t>(A.rows * A.columns * sizeof(double)));
    hipMalloc(&d_B,static_cast<size_t>(B.rows * B.columns * sizeof(double)));
    hipMalloc(&d_C,static_cast<size_t>(C.rows * C.columns * sizeof(double)));

    hipMemcpy(d_A,A.data,static_cast<size_t>(A.rows * A.columns * sizeof(double)),hipMemcpyHostToDevice);
    hipMemcpy(d_B,B.data,static_cast<size_t>(B.rows * B.columns * sizeof(double)),hipMemcpyHostToDevice);

    dim3 blockDim(32,32,1);
    dim3 gridDim((C.columns + 32 -1)/32 + 1,(C.rows + 32 -1)/32 + 1,1);

    matrix_sum_kernel<<<gridDim,blockDim>>>(d_A,d_B,d_C,C.rows,C.columns);

    hipDeviceSynchronize();

    hipMemcpy(C.data,d_C,static_cast<size_t>(C.rows * C.columns * sizeof(double)),hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipDeviceReset();

    return 1;
}
__global__
void matrix_diff_kernel(const double *A,const double *B, double *C,const int C_rows,const int C_columns)
{
    int row    = blockIdx.y * blockDim.y + threadIdx.y; 
    int column = blockIdx.x * blockDim.x + threadIdx.x;

    if(row >= C_rows || column >= C_columns) return;

    C[row * C_columns + column] = A[row*C_columns+column] - B[row*C_columns+column];
}

__host__
int gpu_matrix_diff(const Matrix2d& A,const Matrix2d& B, Matrix2d& C)
{
    if(
        !(            
            A.rows    == B.rows     && 
            A.columns == B.columns  &&
            C.rows    == A.rows     &&
            C.columns == A.columns
         )
      )
    {

        std::cerr << "Given matrices' dimensions don't match:"           << std::endl
                  << "param : A (" << A.rows <<" x " << A.columns << ")" << std::endl
                  << "param : B (" << B.rows <<" x " << B.columns << ")" << std::endl
                  << "param : C (" << C.rows <<" x " << C.columns << ")" << std::endl;
        return -1;
    }

    double *d_A,*d_B,*d_C;

    hipMalloc(&d_A,static_cast<size_t>(A.rows * A.columns * sizeof(double)));
    hipMalloc(&d_B,static_cast<size_t>(B.rows * B.columns * sizeof(double)));
    hipMalloc(&d_C,static_cast<size_t>(C.rows * C.columns * sizeof(double)));

    hipMemcpy(d_A,A.data,static_cast<size_t>(A.rows * A.columns * sizeof(double)),hipMemcpyHostToDevice);
    hipMemcpy(d_B,B.data,static_cast<size_t>(B.rows * B.columns * sizeof(double)),hipMemcpyHostToDevice);

    dim3 blockDim(32,32,1);
    dim3 gridDim((C.columns + 32 -1)/32 + 1,(C.rows + 32 -1)/32 + 1,1);

    matrix_diff_kernel<<<gridDim,blockDim>>>(d_A,d_B,d_C,C.rows,C.columns);

    hipDeviceSynchronize();

    hipMemcpy(C.data,d_C,static_cast<size_t>(C.rows * C.columns * sizeof(double)),hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipDeviceReset();

    return 1;
}